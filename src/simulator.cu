#include "hip/hip_runtime.h"
// Copyright (C) 2022 Codeplay Software Limited
// This work is licensed under the terms of the MIT license.
// For a copy, see https://opensource.org/licenses/MIT.

#include "simulator.cuh"
//#include <cstddef>
#include <stdio.h>

#include <algorithm>
#include <cmath>
#include <random>
#include <tuple>
#include <chrono>
#include <iostream>

namespace simulation {

  // Forward decl
  __global__ void particle_interaction_b(ParticleData_d pPos,
      ParticleData_d pNextPos,
      ParticleData_d pVel, SimParam params);

  __global__ void particle_interaction_nb(ParticleData_d pPos,
      ParticleData_d pNextPos,
      ParticleData_d pVel, SimParam params);

  DiskGalaxySimulator::DiskGalaxySimulator(SimParam params_)
    : params(params_),
    pos(params_.numParticles),
    vel(params_.numParticles),
    pos_d(params_.numParticles),
    vel_d(params_.numParticles),
    pos_next_d(params_.numParticles) {
      randomParticlePos();
      initialParticleVel();
      sendToDevice();
    };

  const std::string* DiskGalaxySimulator::getDeviceName() {
    // Query the device first time only
    if(devName.empty()){
      char devNameHolder[256];
      int error_id = hipDeviceGetName(devNameHolder, 256, 0); // Assume main device
      if(error_id != hipSuccess) devName = "Unknown Device";
      else devName = devNameHolder;
    }
    return &devName;
  }

  void DiskGalaxySimulator::stepSim() {
    // Compute updated positions
    int wg_size = getGwSize();
    int nblocks = ((getNumParticles() - 1) / wg_size) + 1;

    // Profiling info - rather than using the CUDA event recording
    // approach, we are instead measuring the time from before kernel
    // submission until host synchronization. This is more portable via
    // dpct.
    auto start = std::chrono::steady_clock::now();
    for (size_t i = 0; i < params.simIterationsPerFrame; i++) {
      if ( getUseBranch() ) {
        particle_interaction_b<<<nblocks, wg_size>>>(pos_d, pos_next_d, vel_d,
            params);
      } else {
        particle_interaction_nb<<<nblocks, wg_size>>>(pos_d, pos_next_d, vel_d,
            params);
      }
      std::swap(pos_d, pos_next_d);
    }
    gpuErrchk(hipDeviceSynchronize());
    auto stop = std::chrono::steady_clock::now();
    lastStepTime =
      std::chrono::duration<float, std::milli>(stop - start)
      .count();

    // Sync data
    recvFromDevice();
  }

  // Only necessary because we can't initialize data on device yet, in a
  // dpct-friendly way
  void DiskGalaxySimulator::sendToDevice() {
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(pos_d.x, pos.x.data(),
          params.numParticles * sizeof(coords_t),
          hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(pos_d.y, pos.y.data(),
          params.numParticles * sizeof(coords_t),
          hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(pos_d.z, pos.z.data(),
          params.numParticles * sizeof(coords_t),
          hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(vel_d.x, vel.x.data(),
          params.numParticles * sizeof(coords_t),
          hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(vel_d.y, vel.y.data(),
          params.numParticles * sizeof(coords_t),
          hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(vel_d.z, vel.z.data(),
          params.numParticles * sizeof(coords_t),
          hipMemcpyHostToDevice));

    gpuErrchk(hipDeviceSynchronize());
  }

  // Receive particle positions & velocity from device
  void DiskGalaxySimulator::recvFromDevice() {
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(pos.x.data(), pos_d.x,
          params.numParticles * sizeof(coords_t),
          hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(pos.y.data(), pos_d.y,
          params.numParticles * sizeof(coords_t),
          hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(pos.z.data(), pos_d.z,
          params.numParticles * sizeof(coords_t),
          hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(vel.x.data(), vel_d.x,
          params.numParticles * sizeof(coords_t),
          hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(vel.y.data(), vel_d.y,
          params.numParticles * sizeof(coords_t),
          hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(vel.z.data(), vel_d.z,
          params.numParticles * sizeof(coords_t),
          hipMemcpyDeviceToHost));
    gpuErrchk(hipDeviceSynchronize());
  }

  void DiskGalaxySimulator::randomParticlePos() {
    // deterministic - default seed
    std::mt19937 gen;
    std::uniform_real_distribution<> dis(0.0, 1.0);

    // Disk shape in x-y plane
    for (int i = 0; i < params.numParticles; i++) {
      float t = dis(gen) * 2 * PI;
      float s = dis(gen) * 100;
      pos.x[i] = cos(t) * s;
      pos.y[i] = sin(t) * s;
    }

    // Z component is independent (uniform range 0-4)
    std::generate(begin(pos.z), end(pos.z),
        [&gen, &dis]() { return 4.0 * dis(gen); });
  }

  void DiskGalaxySimulator::initialParticleVel() {
    for (int i = 0; i < params.numParticles; i++) {
      vec3 vel = cross({pos.x[i], pos.y[i], pos.z[i]}, {0.0, 0.0, 1.0});
      coords_t orbital_vel = std::sqrt(2.0 * length(vel));
      vel = normalize(vel) * orbital_vel;
      this->vel.x[i] = vel.x;
      this->vel.y[i] = vel.y;
      this->vel.z[i] = vel.z;
    }
  }

  const ParticleData& DiskGalaxySimulator::getParticlePos() { return pos; };

  const ParticleData& DiskGalaxySimulator::getParticleVel() { return vel; };

  // Linear Algebra functions (not yet exposed in header)
  HOSTDEV vec3 cross(const vec3 v0, const vec3 v1) {
    return vec3(v0.y * v1.z - v0.z * v1.y, v0.z * v1.x - v0.x * v1.z,
        v0.x * v1.y - v0.y * v1.x);
  };

  HOSTDEV coords_t length(const vec3 v) {
    return std::sqrt(std::pow(v.x, 2) + std::pow(v.y, 2) + std::pow(v.z, 2));
  }

  HOSTDEV vec3 normalize(const vec3 v) {
    vec3 result = v;
    coords_t len = length(v);
    result.x /= len;
    result.y /= len;
    result.z /= len;
    return result;
  }

  /* O(n^2) implementation (no distance threshold), with no shared
     memory etc.
   */
  __global__ void particle_interaction_b(ParticleData_d pPos,
      ParticleData_d pNextPos,
      ParticleData_d pVel, SimParam params) {
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    if (id >= params.numParticles) return;

    vec3 force(0.0f, 0.0f, 0.0f);
    vec3 pos(pPos.x[id], pPos.y[id], pPos.z[id]);

#pragma unroll 4
    for (int i = 0; i < params.numParticles; i++) {
      vec3 other_pos{pPos.x[i], pPos.y[i], pPos.z[i]};
      vec3 r = other_pos - pos;
      // Fast computation of 1/(|r|^3)
      coords_t dist_sqr = dot(r, r) + params.distEps;
      coords_t inv_dist_cube = rsqrt(dist_sqr * dist_sqr * dist_sqr);

      // assume uniform unit mass
      if ( i == id ) continue;

      force += r * inv_dist_cube;
      //         force += r * inv_dist_cube * (i == id);
    }

    // Update velocity
    vec3 curr_vel(pVel.x[id], pVel.y[id], pVel.z[id]);
    curr_vel *= params.damping;
    curr_vel += force * params.dt * params.G;

    pVel.x[id] = curr_vel.x;
    pVel.y[id] = curr_vel.y;
    pVel.z[id] = curr_vel.z;

    // Update position (integration)
    vec3 curr_pos(pPos.x[id], pPos.y[id], pPos.z[id]);

    curr_pos += curr_vel * params.dt;
    pNextPos.x[id] = curr_pos.x;
    pNextPos.y[id] = curr_pos.y;
    pNextPos.z[id] = curr_pos.z;
  }

  __global__ void particle_interaction_nb(ParticleData_d pPos,
      ParticleData_d pNextPos,
      ParticleData_d pVel, SimParam params) {
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    if (id >= params.numParticles) return;

    vec3 force(0.0f, 0.0f, 0.0f);
    vec3 pos(pPos.x[id], pPos.y[id], pPos.z[id]);

#pragma unroll 4
    for (int i = 0; i < params.numParticles; i++) {
      vec3 other_pos{pPos.x[i], pPos.y[i], pPos.z[i]};
      vec3 r = other_pos - pos;
      // Fast computation of 1/(|r|^3)
      coords_t dist_sqr = dot(r, r) + params.distEps;
      coords_t inv_dist_cube = rsqrt(dist_sqr * dist_sqr * dist_sqr);

      // assume uniform unit mass
      force += r * inv_dist_cube * (i == id);
    }

    // Update velocity
    vec3 curr_vel(pVel.x[id], pVel.y[id], pVel.z[id]);
    curr_vel *= params.damping;
    curr_vel += force * params.dt * params.G;

    pVel.x[id] = curr_vel.x;
    pVel.y[id] = curr_vel.y;
    pVel.z[id] = curr_vel.z;

    // Update position (integration)
    vec3 curr_pos(pPos.x[id], pPos.y[id], pPos.z[id]);

    curr_pos += curr_vel * params.dt;
    pNextPos.x[id] = curr_pos.x;
    pNextPos.y[id] = curr_pos.y;
    pNextPos.z[id] = curr_pos.z;
  }
}  // namespace simulation
